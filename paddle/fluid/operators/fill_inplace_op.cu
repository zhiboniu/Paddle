#include "hip/hip_runtime.h"
/* Copyright (c) 2021 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include <vector>
#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"
#include "paddle/fluid/memory/malloc.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;
using CUDADeviceContext = paddle::platform::CUDADeviceContext;

template <typename T>
__global__ void fill_constant_kernel(const int64_t featuresize, T* in_data,
                                     const T* value) {
  for (int idx = blockIdx.x * featuresize + threadIdx.x;
       idx < (blockIdx.x + 1) * featuresize; idx += blockDim.x) {
    in_data[idx] = *value;
  }
}

template <typename T>
class Fill_CUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& ctx) const override {
#ifdef __HIPCC__
    const int64_t kMaxBlockDim = 256;
#else
    const int64_t kMaxBlockDim = 512;
#endif

    auto* in = ctx.Input<Tensor>("X");
    auto* value = ctx.Input<Tensor>("value");

    auto* in_data = in->data<T>();
    const auto x_dims = in->dims();
    auto matrix_dim = framework::flatten_to_2d(x_dims, 1);
    int64_t batchsize = static_cast<int64_t>(matrix_dim[0]);
    int64_t featuresize = static_cast<int64_t>(matrix_dim[1]);
    int64_t kBlockDim = std::min(featuresize, kMaxBlockDim);
    fill_constant_kernel<T><<<batchsize, kBlockDim, 0>>>(
        featuresize, const_cast<T*>(in_data), value->data<T>());
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
namespace plat = paddle::platform;

REGISTER_OP_CUDA_KERNEL(fill_inplace, ops::Fill_CUDAKernel<float>,
                        ops::Fill_CUDAKernel<double>,
                        ops::Fill_CUDAKernel<plat::float16>,
                        ops::Fill_CUDAKernel<int>,
                        ops::Fill_CUDAKernel<int64_t>,
                        ops::Fill_CUDAKernel<bool>);
